#include "hip/hip_runtime.h"
#include "mat.h"

__global__ void cuda_add(float* a, const float* b, const int column){
    // Single Block
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int index = i * column + j;
    a[index] += b[index];
}

__global__ void cuda_sub(float* a, const float* b, const int column){
    // Single Block
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int index = i * column + j;
    a[index] -= b[index];
}

__global__ void cuda_mul(const float* a, const float* b, float* result, const int row, const int column){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float tmpValue = 0.0;
    for (int k = 0; k < row; ++k){
        tmpValue += a[i * column + k] * b[k * row + j];
    }
    result[i * row + j] = tmpValue;
}

__global__ void cuda_mul_shared(const float* a, const float* b, float* result, const int row, const int column)
{
    __shared__ float Mds[1][1];
    __shared__ float Nds[1][1];

}

void Mat::add(const Mat& m){
    dimensionCheck(m);
    CUDA_CHECK(hipSetDevice(DEVICE));
    size_t bytesize = sizeof(float) * column * row;
    float* d_a = NULL;
    float* d_b = NULL;
    CUDA_CHECK(hipMalloc((void**) &d_a, bytesize));
    CUDA_CHECK(hipMalloc((void**) &d_b, bytesize));
    CUDA_CHECK(hipMemcpy(d_a, matrix, bytesize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, m.get_mat(), bytesize, hipMemcpyHostToDevice));

    dim3 blockPerGrid (1, 1);
    dim3 threadPerGrid (row, column);
    cuda_add<<< blockPerGrid, threadPerGrid >>>(d_a, d_b, column);

    CUDA_CHECK(hipMemcpy(matrix, d_a, bytesize, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipDeviceReset());
}

void Mat::sub(const Mat& m){
    dimensionCheck(m);
    CUDA_CHECK(hipSetDevice(DEVICE));
    size_t bytesize = sizeof(float) * column * row;
    float* d_a = NULL;
    float* d_b = NULL;
    CUDA_CHECK(hipMalloc((void**) &d_a, bytesize));
    CUDA_CHECK(hipMalloc((void**) &d_b, bytesize));
    CUDA_CHECK(hipMemcpy(d_a, matrix, bytesize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, m.get_mat(), bytesize, hipMemcpyHostToDevice));

    dim3 blockPerGrid (1, 1);
    dim3 threadPerGrid (row, column);
    cuda_sub<<< blockPerGrid, threadPerGrid >>>(d_a, d_b, column);

    CUDA_CHECK(hipMemcpy(matrix, d_a, bytesize, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipDeviceReset());
}

void Mat::mul(const Mat &m){
    // dimesion check
    if (row != m.get_column() || column != m.get_row()){
        std::cerr << "Matrix dimension error: (" << row << ", " << column << ") cannot match (" << m.get_row() << ", " << m.get_column() << ")" << std::endl;
        exit(-1);
    }
    float* result = new float[row * row];
    CUDA_CHECK(hipSetDevice(DEVICE));
    size_t bytesize = sizeof(float) * column * row;
    size_t bytesize_result = sizeof(float) * row * row;
    float* d_a = NULL;
    float* d_b = NULL;
    float* d_result = NULL;
    CUDA_CHECK(hipMalloc((void **) &d_a, bytesize));
    CUDA_CHECK(hipMalloc((void **) &d_b, bytesize));
    CUDA_CHECK(hipMalloc((void **) &d_result, bytesize_result));

    CUDA_CHECK(hipMemcpy(d_a, matrix, bytesize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, m.get_mat(), bytesize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_result, result, bytesize_result, hipMemcpyHostToDevice));

    dim3 blockPerGrid (1, 1);
    dim3 threadPerBlock (row, row);
    cuda_mul<<<blockPerGrid, threadPerBlock>>>(d_a, d_b, d_result, row, column);

    CUDA_CHECK(hipMemcpy(result, d_result, bytesize_result, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_result));
    delete[] matrix;
    matrix = result;
    column = row;
}

void Mat::copy(const Mat &m){
    dimensionCheck(m);
    matrix = new float[row * column];
    CUDA_CHECK(hipSetDevice(DEVICE));
    size_t bytesize = sizeof(float) * column * row;
    float* d_tmp = NULL;
    CUDA_CHECK(hipMalloc((void**) &d_tmp, bytesize));
    CUDA_CHECK(hipMemcpy(d_tmp, m.get_mat(), bytesize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(matrix, d_tmp, bytesize, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_tmp));
}

void Mat::information(bool verbose){
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, DEVICE));
    maxThreadsPerBlock = prop.maxThreadsPerBlock;
    maxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
    if (verbose){
        printf("Name: %s\n", prop.name);
        printf("sharedMemPerBlock: %lu kB\n", prop.sharedMemPerBlock / 1024lu); // in byte
        printf("warpSize: %d threads\n", prop.warpSize);
        printf("maxThreadsPerBlock: %d\n", maxThreadsPerBlock);
        printf("maxThreadsPerMultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
        printf("maxThreadsDim: %d, %d, %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    }
}
    